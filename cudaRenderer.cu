#include "hip/hip_runtime.h"
#include <algorithm>
#include <string>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "util.h"

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {

    int numberOfTrees;

    float *position;
    float *color;
    LSystem *trees;

    int imageWidth;
    int imageHeight;
    float *imageData;
};

// Global variable that is in scope, but read-only, for all CUDA
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (We didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;



// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}


////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
    image = NULL;

    numberOfTrees = 0;
    position = NULL;
    color = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {
    if (image) {
        delete image;
    }

    if (position) {
        delete[] position;
        delete[] color;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceImageData);
    }
}

const Image *CudaRenderer::getImage() {
    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data, cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void CudaRenderer::loadTrees(LSystem *trees, int numberOfTrees) {
    this->trees = trees;
    this->numberOfTrees = numberOfTrees;
}

void CudaRenderer::setup() {
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0) {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU) {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    //TODO: Copy lines over correctly
    int numberOfLines = trees[0].numLines();
    hipMalloc(&cudaDevicePosition, sizeof(float) * 4 * numberOfTrees * numberOfLines);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfTrees);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    for (int i = 0 ; i < numberOfTrees ; i++) {
        hipMemcpy(cudaDevicePosition + sizeof(float)*4*numberOfLines*i, trees[i].lines, sizeof(float) * 4 * numberOfLines, hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceColor + sizeof(float)*3*i, trees[i].color, sizeof(float) * 3, hipMemcpyHostToDevice);
    }
    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.numberOfTrees = numberOfTrees;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.color = cudaDeviceColor;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

}

// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height) {
    if (image)
        delete image;
    image = new Image(width, height);
}

// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
                 (image->height + blockDim.y - 1) / blockDim.y);

    kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    hipDeviceSynchronize();
}

// Does nothing
void CudaRenderer::advanceAnimation() {
    return;
}

void CudaRenderer::render() {
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numberOfTrees + blockDim.x - 1) / blockDim.x);

    // TODO: Render lines
    // kernelRenderCircles<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
}
