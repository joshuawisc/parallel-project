#include "hip/hip_runtime.h"
#include <algorithm>
#include <string>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "util.h"
#include "image.h"
#include "cycleTimer.h"

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {

    int numberOfTrees;
    int numberOfLines;

    float *position;
    float *color;
    LSystem *trees;

    int imageWidth;
    int imageHeight;
    float *imageData;
};

// Global variable that is in scope, but read-only, for all CUDA
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (We didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;



// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

__device__ void drawLine(float x0, float y0, float x1, float y1, float r, float g, float b) {
    // Can't do (1.0)??
    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    x1 = min(max(static_cast<int>(x1 * width), 0), width-1);
    y1 = min(max(static_cast<int>(y1 * height), 0), height-1);
    x0 = min(max(static_cast<int>(x0 * width), 0), width-1);
    y0 = min(max(static_cast<int>(y0 * height), 0), height-1);
    int dx = abs(x1 - x0);
    int sx = x0 < x1 ? 1 : -1;
    int dy = -abs(y1 - y0);
    int sy = y0 < y1 ? 1 : -1;
    int error = dx + dy;
    int e2;

    while (true) {
        // plot(x0, y0);
        // TODO: USE FLOAT4 INSTEAD
        float *imgPtr = &cuConstRendererParams.imageData[4 * (int(y0) * width + int(x0))];
        imgPtr[0] = r;
        imgPtr[1] = g;
        imgPtr[2] = b;
        imgPtr[3] = 1;

        if (x0 == x1 && y0 == y1)
            break;
        e2 = 2 * error;
        if (e2 >= dy) {
            if (x0 == x1)
                break;
            error = error + dy;
            x0 = x0 + sx;
        }
        if (e2 <= dx) {
            if (y0 == y1)
                break;
            error = error + dx;
            y0 = y0 + sy;
        }
    }
}

__global__ void kernelRenderLines() {
    GlobalConstants params = cuConstRendererParams;
    int numberOfLines = params.numberOfLines;
    int numberOfTrees = params.numberOfTrees;
    int lineIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int treeIndex = lineIndex / numberOfLines;
    if (lineIndex >= numberOfLines*numberOfTrees)
        return;
    lineIndex *= 4;
    treeIndex *= 3;
    drawLine(params.position[lineIndex], params.position[lineIndex+1], params.position[lineIndex+2], params.position[lineIndex+3],
        params.color[treeIndex], params.color[treeIndex+1], params.color[treeIndex+2]);

}


////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer() {
    image = NULL;

    numberOfTrees = 0;
    numberOfLines = 0;
    position = NULL;
    color = NULL;
    trees = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {
    if (image) {
        delete image;
    }

    if (position) {
        delete[] position;
        delete[] color;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceImageData);
    }
}

const Image *CudaRenderer::getImage() {
    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    // printf("Copying image data from device\n");
    
    double preSetup = CycleTimer::currentSeconds();
    hipMemcpy(image->data, cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    double postSetup = CycleTimer::currentSeconds();
    // printf("CUDA getImage time:\t%.3f ms\n", 1000.f * (postSetup - preSetup));
    return image;
}

void CudaRenderer::loadTrees(LSystem *trees, int numberOfTrees) {
    this->trees = trees;
    this->numberOfTrees = numberOfTrees;
}

void CudaRenderer::setup(int threads) {
    double preSetup = CycleTimer::currentSeconds();
    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("NVIDIA GeForce RTX 2080") == 0) {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU) {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    //TODO: Copy lines over correctly
    this->numberOfLines = trees[0].numLines(trees[0].depth);
    hipMalloc(&cudaDevicePosition, sizeof(float) * 4 * numberOfTrees * numberOfLines);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfTrees);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    for (int i = 0 ; i < numberOfTrees ; i++) {
        hipMemcpy(cudaDevicePosition + 4*numberOfLines*i, &trees[i].lines.front(), sizeof(float) * 4 * numberOfLines, hipMemcpyHostToDevice);
        hipMemcpy(cudaDeviceColor + 3*i, trees[i].color, sizeof(float) * 3, hipMemcpyHostToDevice);
    }
    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.numberOfTrees = numberOfTrees;
    params.numberOfLines = numberOfLines;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.color = cudaDeviceColor;
    params.imageData = cudaDeviceImageData;
    params.trees = trees;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));
    double postSetup = CycleTimer::currentSeconds();
    printf("CUDA setup time:\t%.3f ms\n", 1000.f * (postSetup - preSetup));

}

// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height) {
    if (image)
        delete image;
    image = new Image(width, height);
}

// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage() {
    double preClear = CycleTimer::currentSeconds();
    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
                 (image->height + blockDim.y - 1) / blockDim.y);

    kernelClearImage<<<gridDim, blockDim>>>(0.f, 0.f, 0.f, 1.f);
    hipDeviceSynchronize();
    double postClear = CycleTimer::currentSeconds();
    printf("CUDA clear time:\t%.3f ms\n", 1000.f * (postClear - preClear));
}

// Does nothing
void CudaRenderer::advanceAnimation() {
    return;
}

void CudaRenderer::render() {
    // 256 threads per block is a healthy number
    dim3 blockDim(1024, 1);
    dim3 gridDim((numberOfTrees*numberOfLines + blockDim.x - 1) / blockDim.x);
    
    double preRender = CycleTimer::currentSeconds();
    // TODO: Render lines
    kernelRenderLines<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    double postRender = CycleTimer::currentSeconds();
    printf("CUDA render time:\t%.3f ms\n", 1000.f * (postRender - preRender));
}
